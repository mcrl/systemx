#include "hip/hip_runtime.h"
// Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.

#include "spdlog/spdlog.h"
#include "hip/hip_runtime.h"

#include "driver.hpp"
#include "kernels.hpp"
#include "utils.hpp"

#define PCIE_STORE_NUM_FLOATS_PER_STEP 800000000 // 3.2GB
#define PCIE_READ_NUM_FLOATS_PER_STEP 800000000 // 3.2GB
#define P2P true // TODO: make this as a kargs option 
#define TRANSFER_TYPE float4

typedef enum {
  CE = 0,
  SM = 1,
} P2PEngine;

P2PEngine p2p_mechanism = SM; // By default use SM initiated p2p transfers
                              // TODO: add support for CE initiated p2p transfers
using SYSTEMX::core::Driver;

// This kernel is for demonstration purposes only, not a performant kernel for p2p transfers.
// num_elems is the number of T in dest (and src)
template<typename T>
__global__ void copyp2p_kernel(T *__restrict__ dest, T const *__restrict__ src,
                               size_t num_elems, const uint steps) {
  size_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = gridDim.x * blockDim.x;

#pragma unroll
  for (uint i = 0; i < steps; i++) {
#pragma unroll(5)
    for (size_t j = globalId; j < num_elems; j += stride) {
      dest[j] = src[j];
    }
  }
}

void Driver::pcieReadRun(kernel_run_args *args) {
  spdlog::trace(__PRETTY_FUNCTION__);

  assertDeviceCorrect();

  // create event to check if buffer is ready
  hipEvent_t ready;
  CUDA_CALL(hipEventCreate(&ready));
  
  // set appropriate device buffer
  const int in_size = PCIE_READ_NUM_FLOATS_PER_STEP;  
  CUDA_CALL(hipMallocAsync(&((*((args->shared_buffer_map)->at("d_in")))[gpu_index_]),
                            in_size * sizeof(float),
                            args->stream));
  CUDA_CALL(hipEventRecord(ready, args->stream));

  // check if all buffers are ready
  CUDA_CALL(hipEventSynchronize(ready));

  // check if all gpus are ready
  (*(args->shared_counter_map))["deviceBufferReady"]->decrement();

  // start kernel
  hipEvent_t start, end;
  start = std::get<1>(args->events[0]);
  end = std::get<1>(args->events[1]);

  float elapsed_ms;
  CUDA_CALL(hipEventRecord(start, args->stream));

  // work per thread: float4 * (in_size / 4 / (dimGrid.x * dimBlock.x))
  const int intra_step_access_per_thread = in_size / (args->dimGrid.x * args->dimBlock.x);

  // TODO: make multiple srcs possible
  int access = 0;
  CUDA_CALL(hipDeviceCanAccessPeer(&access, gpu_index_, gpu_index_ == 0 ? 1 : 0));
  
  if (P2P && access && p2p_mechanism == SM) {
    copyp2p_kernel<TRANSFER_TYPE> << <args->dimGrid, args->dimBlock, 0, args->stream >> > (
      (TRANSFER_TYPE *)(*((args->shared_buffer_map)->at("d_in")))[gpu_index_],
      (TRANSFER_TYPE *)(*((args->shared_buffer_map)->at("d_in")))[gpu_index_ == 0 ? 1 : 0],
      in_size / (sizeof(TRANSFER_TYPE) / sizeof(float)), args->steps);
  } else {
    hipMemcpyPeerAsync(
      (*((args->shared_buffer_map)->at("d_in")))[gpu_index_], gpu_index_,
      (*((args->shared_buffer_map)->at("d_in")))[gpu_index_ == 0 ? 1 : 0], gpu_index_ == 0 ? 1 : 0,
      in_size * sizeof(float), args->stream);
  }
  CUDA_CALL(hipEventRecord(end, args->stream));
  CUDA_CALL(hipEventSynchronize(end));
  CUDA_CALL(hipEventElapsedTime(&elapsed_ms, start, end));

  const int total_threads = get_nthreads(args->dimGrid, args->dimBlock);
  double per_thread_bandwidth = args->steps * intra_step_access_per_thread * sizeof(float) / elapsed_ms / 1e6;
  double bandwidth = per_thread_bandwidth * total_threads;
  spdlog::info("{}(id: {}) {:.2f} GB/s {:d} ms", FUNC_NAME(copyp2p_kernel), args->id, bandwidth, (int)elapsed_ms);

  // check if all gpus are finished
  (*(args->shared_counter_map))["deviceKernelFinish"]->decrement();
  
  // cleanup
  CUDA_CALL(hipFree((*((args->shared_buffer_map)->at("d_in")))[gpu_index_]));
  CUDA_CALL(hipEventDestroy(ready));
  CUDA_CALL(hipEventDestroy(start));
  CUDA_CALL(hipEventDestroy(end));
}

void Driver::pcieWriteRun(kernel_run_args *args) {
  spdlog::trace(__PRETTY_FUNCTION__);

  assertDeviceCorrect();

  // create event to check if buffer is ready
  hipEvent_t ready;
  CUDA_CALL(hipEventCreate(&ready));
  
  // set appropriate device buffer
  const int in_size = PCIE_STORE_NUM_FLOATS_PER_STEP;  
  CUDA_CALL(hipMallocAsync(&((*((args->shared_buffer_map)->at("d_in")))[gpu_index_]),
                            in_size * sizeof(float),
                            args->stream));
  CUDA_CALL(hipEventRecord(ready, args->stream));

  // check if all buffers are ready
  CUDA_CALL(hipEventSynchronize(ready));

  // check if all gpus are ready
  (*(args->shared_counter_map))["deviceBufferReady"]->decrement();

  // start kernel
  hipEvent_t start, end;
  start = std::get<1>(args->events[0]);
  end = std::get<1>(args->events[1]);

  float elapsed_ms;
  CUDA_CALL(hipEventRecord(start, args->stream));

  // work per thread: float4 * (in_size / 4 / (dimGrid.x * dimBlock.x))
  const int intra_step_access_per_thread = in_size / (args->dimGrid.x * args->dimBlock.x);

  // TODO: make multiple dests possible
  int access = 0;
  CUDA_CALL(hipDeviceCanAccessPeer(&access, gpu_index_, gpu_index_ == 0 ? 1 : 0));
  if (P2P && access && p2p_mechanism == SM) {
    copyp2p_kernel<TRANSFER_TYPE> << <args->dimGrid, args->dimBlock, 0, args->stream >> > (
      (TRANSFER_TYPE *)(*((args->shared_buffer_map)->at("d_in")))[gpu_index_ == 0 ? 1 : 0],
      (TRANSFER_TYPE *)(*((args->shared_buffer_map)->at("d_in")))[gpu_index_],
      in_size / (sizeof(TRANSFER_TYPE) / sizeof(float)), args->steps);
  } else {
    hipMemcpyPeerAsync(
      (*((args->shared_buffer_map)->at("d_in")))[gpu_index_ == 0 ? 1 : 0], gpu_index_ == 0 ? 1 : 0,
      (*((args->shared_buffer_map)->at("d_in")))[gpu_index_], gpu_index_,
      in_size * sizeof(float), args->stream);
  }
  CUDA_CALL(hipEventRecord(end, args->stream));
  CUDA_CALL(hipEventSynchronize(end));
  CUDA_CALL(hipEventElapsedTime(&elapsed_ms, start, end));

  const int total_threads = get_nthreads(args->dimGrid, args->dimBlock);
  double per_thread_bandwidth = args->steps * intra_step_access_per_thread * sizeof(float) / elapsed_ms / 1e6;
  double bandwidth = per_thread_bandwidth * total_threads;
  spdlog::info("{}(id: {}) {:.2f} GB/s {:d} ms", FUNC_NAME(copyp2p_kernel), args->id, bandwidth, (int)elapsed_ms);

  // check if all gpus are finished
  (*(args->shared_counter_map))["deviceKernelFinish"]->decrement();
  
  // cleanup
  CUDA_CALL(hipFree((*((args->shared_buffer_map)->at("d_in")))[gpu_index_]));
  CUDA_CALL(hipEventDestroy(ready));
  CUDA_CALL(hipEventDestroy(start));
  CUDA_CALL(hipEventDestroy(end));
}