#include "spdlog/spdlog.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "driver.hpp"
#include "kernels.hpp"
#include "utils.hpp"

using SYSTEMX::core::Driver;

// TODO: Refactor
void Driver::cublasGemmRun(kernel_run_args *args) {
  spdlog::trace(__PRETTY_FUNCTION__);

  hipStream_t stream = args->stream;
  hipblasHandle_t handle;
  CUBLAS_CALL(hipblasCreate(&handle));
  CUBLAS_CALL(hipblasSetStream(handle, stream));

  float *d_A, *d_B, *d_C;
  int M = args->dimGrid.x * args->dimGrid.y * args->dimGrid.z * args->dimBlock.x * args->dimBlock.y * args->dimBlock.z,
    K = args->dimGrid.x * args->dimGrid.y * args->dimGrid.z * args->dimBlock.x * args->dimBlock.y * args->dimBlock.z,
    N = args->dimGrid.x * args->dimGrid.y * args->dimGrid.z * args->dimBlock.x * args->dimBlock.y * args->dimBlock.z;

  CUDA_CALL(hipMallocAsync(&d_A, M * K * sizeof(float), stream));
  CUDA_CALL(hipMallocAsync(&d_B, K * N * sizeof(float), stream));
  CUDA_CALL(hipMallocAsync(&d_C, M * N * sizeof(float), stream));

  CUDA_CALL(hipMemsetAsync(d_A, 1, M * K * sizeof(float), stream));
  CUDA_CALL(hipMemsetAsync(d_B, 1, K * N * sizeof(float), stream));
  CUDA_CALL(hipMemsetAsync(d_C, 0, M * N * sizeof(float), stream));
    
  // Launch non-blocking compute
  const float alpha = 1.0f;
  const float beta  = 0.0f;
  // CUBLAS_CALL(cublasSetSmCountTarget(handle, 60)); // set cublas SM count

  hipEvent_t start, end;
  start = std::get<1>(args->events[0]);
  end = std::get<1>(args->events[1]);

  CUDA_CALL(hipEventRecord(start, args->stream));
  CUBLAS_CALL(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N));
  CUDA_CALL(hipEventRecord(end, args->stream));

  float elapsed_time = 0;
  CUDA_CALL(hipEventSynchronize(end));
  CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, end));
  spdlog::info("Kernel {} took {} ms", FUNC_NAME(idle_kernel), elapsed_time);
  
  // cleanup
  CUBLAS_CALL(hipblasDestroy(handle));
}