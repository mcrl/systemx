#include "spdlog/spdlog.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "driver.hpp"
#include "kernels.hpp"
#include "utils.hpp"

using SYSTEMX::core::Driver;

// TODO: Refactor
void Driver::cublasGemmRun(kernel_run_args *args) {
  spdlog::trace(__PRETTY_FUNCTION__);

  hipStream_t stream = args->stream;

  hipblasHandle_t handle = createCublasHandle();
  CUBLAS_CALL(hipblasSetStream(handle, stream));

  float *d_A, *d_B, *d_C;
  int M = 8192, K = 8192, N = 8192;

  CUDA_CALL(hipMallocAsync(&d_A, M * K * sizeof(float), stream));
  CUDA_CALL(hipMallocAsync(&d_B, K * N * sizeof(float), stream));
  CUDA_CALL(hipMallocAsync(&d_C, M * N * sizeof(float), stream));

  CUDA_CALL(hipMemsetAsync(d_A, 1, M * K * sizeof(float), stream));
  CUDA_CALL(hipMemsetAsync(d_B, 1, K * N * sizeof(float), stream));
  CUDA_CALL(hipMemsetAsync(d_C, 0, M * N * sizeof(float), stream));
    
  // Launch non-blocking compute
  const float alpha = 1.0f;
  const float beta  = 0.0f;
  // CUBLAS_CALL(cublasSetSmCountTarget(handle, 60)); // set cublas SM count
  CUBLAS_CALL(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N));
}