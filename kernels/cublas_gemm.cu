#include "spdlog/spdlog.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "driver.hpp"
#include "kernels.hpp"
#include "utils.hpp"

using SYSTEMX::core::Driver;

void Driver::cublasGemmRun() {
  spdlog::trace(__PRETTY_FUNCTION__);

  hipStream_t stream = createStream();

  hipblasHandle_t handle = createCublasHandle();
  CUBLAS_CALL(hipblasSetStream(handle, stream));

  float *d_A, *d_B, *d_C;
  int M = 8192, K = 8192, N = 8192;

  CUDA_CALL(hipMallocAsync(&d_A, M * K * sizeof(float), stream));
  CUDA_CALL(hipMallocAsync(&d_B, K * N * sizeof(float), stream));
  CUDA_CALL(hipMallocAsync(&d_C, M * N * sizeof(float), stream));

  CUDA_CALL(hipMemsetAsync(d_A, 1, M * K * sizeof(float), stream));
  CUDA_CALL(hipMemsetAsync(d_B, 1, K * N * sizeof(float), stream));
  CUDA_CALL(hipMemsetAsync(d_C, 0, M * N * sizeof(float), stream));
    
  // Launch non-blocking compute
  const float alpha = 1.0f;
  const float beta  = 0.0f;
  // CUBLAS_CALL(cublasSetSmCountTarget(handle, 60)); // set cublas SM count
  CUBLAS_CALL(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N));
}