#include "spdlog/spdlog.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "driver.hpp"
#include "kernels.hpp"

using SYSTEMX::core::Driver;

// TODO: 
//  - Set appropriate dimension sizes and SM count according to args->dimGrid/dimBlock
void Driver::cublasGemmRun(kernel_run_args *args) {
  spdlog::trace(__PRETTY_FUNCTION__);

  hipStream_t stream = args->stream;
  hipblasHandle_t handle;
  CUBLAS_CALL(hipblasCreate(&handle));
  CUBLAS_CALL(hipblasSetStream(handle, stream));

  float *d_A, *d_B, *d_C;
  uint64_t M = 8192, K = 8192, N = 8192;

  CUDA_CALL(hipMallocAsync(&d_A, M * K * sizeof(float), stream));
  CUDA_CALL(hipMallocAsync(&d_B, K * N * sizeof(float), stream));
  CUDA_CALL(hipMallocAsync(&d_C, M * N * sizeof(float), stream));

  CUDA_CALL(hipMemsetAsync(d_A, 1, M * K * sizeof(float), stream));
  CUDA_CALL(hipMemsetAsync(d_B, 1, K * N * sizeof(float), stream));
  CUDA_CALL(hipMemsetAsync(d_C, 0, M * N * sizeof(float), stream));
    
  // Launch non-blocking compute
  const float alpha = 1.0f;
  const float beta  = 0.0f;
  // CUBLAS_CALL(cublasSetSmCountTarget(handle, 60)); // set cublas SM count

  hipEvent_t start, end;
  start = std::get<1>(args->events[0]);
  end = std::get<1>(args->events[1]);

  float elapsed_ms;
  CUDA_CALL(hipEventRecord(start, args->stream));
  CUBLAS_CALL(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N));
  CUDA_CALL(hipEventRecord(end, args->stream));
  CUDA_CALL(hipEventSynchronize(end));
  CUDA_CALL(hipEventElapsedTime(&elapsed_ms, start, end));

  double gflops = 2.0 * M * K * N / elapsed_ms * 1e3 / 1e9;
  spdlog::info("{}(id: {}) {:.2f} Gflops {:d} ms", "cublasGemm", args->id, gflops, elapsed_ms);

  // cleanup
  CUDA_CALL(hipFree(d_A));
  CUDA_CALL(hipFree(d_B));
  CUDA_CALL(hipFree(d_C));
  CUBLAS_CALL(hipblasDestroy(handle));
  CUDA_CALL(hipEventDestroy(start));
  CUDA_CALL(hipEventDestroy(end));
}