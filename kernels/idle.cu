#include "hip/hip_runtime.h"
#include <tuple>

#include "spdlog/spdlog.h"
#include "hip/hip_runtime.h"

#include "driver.hpp"
#include "kernels.hpp"
#include "utils.hpp"

using SYSTEMX::core::Driver;

__global__ void idle_kernel(uint milliseconds) {
  for (uint i = 0; i < milliseconds; ++i) {
    __nanosleep(1000000U);
  }
}

void Driver::idleRun(kernel_run_args *args) {
  spdlog::trace(__PRETTY_FUNCTION__);

  assertDeviceCorrect();

  uint milliseconds = 300;

  hipEvent_t start, end;
  start = std::get<1>(args->events[0]);
  end = std::get<1>(args->events[1]);

  CUDA_CALL(hipEventRecord(start, args->stream));
  idle_kernel << <args->dimGrid, args->dimBlock, 0, args->stream >> > (milliseconds);
  CUDA_CALL(hipEventRecord(end, args->stream));

  float elapsed_ms;
  CUDA_CALL(hipEventSynchronize(end));
  CUDA_CALL(hipEventElapsedTime(&elapsed_ms, start, end));
  spdlog::info("{}(id:{:d}) {:d} ms", FUNC_NAME(idle_kernel), args->id, elapsed_ms);

  // cleanup
  CUDA_CALL(hipEventDestroy(start));
  CUDA_CALL(hipEventDestroy(end));
}