#include "hip/hip_runtime.h"
#include "spdlog/spdlog.h"
#include "hip/hip_runtime.h"

#include "driver.hpp"

using SYSTEMX::core::Driver;

__global__ void idle_kernel(uint seconds) {
  for (uint i = 0; i < seconds; ++i) {
    for (int j = 0; j < 1000; j++) {
      __nanosleep(1000000U);
    }
  }
}

void Driver::idleRun() {
  spdlog::info(__PRETTY_FUNCTION__);

  hipStream_t stream = createStream();

  uint idle_seconds = 10;
  
  dim3 gridDim(1, 1, 1);
  dim3 blockDim(1, 1, 1);
  idle_kernel<<<gridDim, blockDim, 0, stream>>>(idle_seconds);
}