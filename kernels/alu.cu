#include "hip/hip_runtime.h"
#include<cstdlib>
#include<ctime>

#include "spdlog/spdlog.h"
#include "hip/hip_runtime.h"

#include "driver.hpp"
#include "kernels.hpp"
#include "utils.hpp"

#define WPT 8 // Hyperparameter to maximize register spilling of local memory
#define STEPS 12000000

using SYSTEMX::core::Driver;

__global__ void alu_compute_kernel(float *d, const float seed) {  
  float tmps[WPT];
  int id = blockDim.x * blockIdx.x + threadIdx.x;

#pragma unroll
  for (int i = 0; i < WPT; i++) {
    tmps[i] = id / 3.0f; // some random initialization to avoid gmem access
#pragma unroll
    for (int j = 0; j < STEPS; j++) {
      tmps[i] = mad(tmps[i], tmps[i], seed);
    }
  }

  // To avoid kernel optimization
  float sum = int2floatCast(0);
#pragma unroll
  for (int j = 0; j < WPT; j += 2) {
    sum = mad(tmps[j], tmps[j + 1], sum);
    // Never executed, to avoid kernel optimization
    // If not kernel execution is skipped
    if (sum == int2floatCast(-1)) {
      d[id * WPT + j] = sum;
    }    
  }
}

void Driver::aluComputeRun(kernel_run_args *args) {
  spdlog::trace(__PRETTY_FUNCTION__);

  srand((unsigned int)time(NULL));
  const int seed = rand();

  float *d_in;
  CUDA_CALL(hipMallocAsync(&d_in, args->dimGrid.x * args->dimBlock.x * sizeof(float), args->stream)); 

  hipEvent_t start, end;
  start = std::get<1>(args->events[0]);
  end = std::get<1>(args->events[1]);

  float elapsed_ms;
  CUDA_CALL(hipEventRecord(start, args->stream));
  alu_compute_kernel << <args->dimGrid, args->dimBlock, 0, args->stream >> > (d_in, seed);
  CUDA_CALL(hipEventRecord(end, args->stream));
  CUDA_CALL(hipEventSynchronize(end));
  CUDA_CALL(hipEventElapsedTime(&elapsed_ms, start, end));

  const int total_threads = get_nthreads(args->dimGrid, args->dimBlock);
  double gflops = 2.0 * (STEPS * WPT + WPT) * total_threads / elapsed_ms * 1e3 / 1e9;
  spdlog::info("{}(id: {}) {:.2f} Gflops {:d} ms", FUNC_NAME(alu_compute_kernel), args->id, gflops, elapsed_ms);

  // cleanup
  CUDA_CALL(hipFree(d_in));
  CUDA_CALL(hipEventDestroy(start));
  CUDA_CALL(hipEventDestroy(end));
}