#include "hip/hip_runtime.h"
#include "spdlog/spdlog.h"
#include "hip/hip_runtime.h"

#include "driver.hpp"
#include "kernels.hpp"

using SYSTEMX::core::Driver;

__global__ void l2_load_kernel(float *in, const int in_size,
                               const int stride, const uint steps) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  
  // To avoid kernel optimization
  float sum = int2floatCast(0);
  for (uint i = 0; i < steps; i++) {
    int idx = id;
    
    for (uint j = 0; j < in_size / stride; j++) {
      register float tmp;
      // load from gmem bypassing l1 cache
      asm volatile(
        "{\n\t"
        "ld.global.cg.f32 %0, [%1];\n\t"
        "}"
        : "=f"(tmp)
        : "l"(&in[idx])
        : "memory");
      sum += tmp;
      idx = (idx + stride) % in_size;
    }
  }
  
  // fake store to `in` to avoid compiler optimization
  if (sum == int2floatCast(-1)) {
    in[id] = sum;
  }
}

__global__ void l2_store_kernel(float *out, const int out_size,
                                const int stride, const uint steps) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;

  // To avoid kernel optimization
  register float src = int2floatCast(0);
  for (uint i = 0; i < steps; i++) {
    int idx = id;
    
    for (uint j = 0; j < out_size / stride; j++) {
      register float *out_ptr = &out[idx];
      // store to gmem bypassing l1 cache
      asm volatile(
        "{\n\t"
        "st.global.cg.f32 [%0], %1;\n\t"
        "}"
        : "+l"(out_ptr)
        : "f"(src)
        : "memory");
      idx = (idx + stride) % out_size;
    }
  }
}

void Driver::l2LoadRun(kernel_run_args *args) {
  spdlog::trace(__PRETTY_FUNCTION__);

  assertDeviceCorrect();

  const int l2CacheSizeBytes = device_properties_.l2CacheSize;
  const int stride = device_properties_.warpSize; 
  const int in_size = l2CacheSizeBytes / sizeof(float); // in_size is set to L2 cache size, which will make
                                                  // L2 hit rate ~100%
  const int intra_step_access_per_thread = in_size / stride;
  
  float *d_in;
  CUDA_CALL(hipMallocAsync(&d_in, in_size * sizeof(float), args->stream));
  CUDA_CALL(hipMemsetAsync(d_in, 0.0f, in_size * sizeof(float), args->stream));
  
  hipEvent_t start, end;
  start = std::get<1>(args->events[0]);
  end = std::get<1>(args->events[1]);

  float elapsed_ms;
  CUDA_CALL(hipEventRecord(start, args->stream));
  l2_load_kernel << <args->dimGrid, args->dimBlock, 0, args->stream >> > (d_in, in_size, stride, args->steps);
  CUDA_CALL(hipEventRecord(end, args->stream));
  CUDA_CALL(hipEventSynchronize(end));
  CUDA_CALL(hipEventElapsedTime(&elapsed_ms, start, end));

  const int total_threads = get_nthreads(args->dimGrid, args->dimBlock);

  double per_thread_bandwidth = args->steps * intra_step_access_per_thread * sizeof(float) / elapsed_ms / 1e6;
  double bandwidth = per_thread_bandwidth * total_threads;
  spdlog::info("{}(id: {}) {:.2f} GB/s {:d} ms", FUNC_NAME(l2_load_kernel), args->id, bandwidth, (int)elapsed_ms);
  
  // cleanup
  CUDA_CALL(hipFree(d_in));
  CUDA_CALL(hipEventDestroy(start));
  CUDA_CALL(hipEventDestroy(end));
}

void Driver::l2StoreRun(kernel_run_args *args) {
  spdlog::trace(__PRETTY_FUNCTION__);

  assertDeviceCorrect();

  const int l2CacheSizeBytes = device_properties_.l2CacheSize;
  const int stride = device_properties_.warpSize; 
  const int out_size = l2CacheSizeBytes / sizeof(float); // out_size is set to L2 cache size, which will make
                                                  // L2 hit rate ~100%
  const int intra_step_access_per_thread = out_size / stride;
  
  float *d_out;
  CUDA_CALL(hipMallocAsync(&d_out, out_size * sizeof(float), args->stream));
  
  hipEvent_t start, end;
  start = std::get<1>(args->events[0]);
  end = std::get<1>(args->events[1]);

  float elapsed_ms;
  CUDA_CALL(hipEventRecord(start, args->stream));
  l2_store_kernel << <args->dimGrid, args->dimBlock, 0, args->stream >> > (d_out, out_size, stride, args->steps);
  CUDA_CALL(hipEventRecord(end, args->stream));
  CUDA_CALL(hipEventSynchronize(end));
  CUDA_CALL(hipEventElapsedTime(&elapsed_ms, start, end));

  const int total_threads = get_nthreads(args->dimGrid, args->dimBlock);

  double per_thread_bandwidth = args->steps * intra_step_access_per_thread * sizeof(float) / elapsed_ms / 1e6;
  double bandwidth = per_thread_bandwidth * total_threads;
  spdlog::info("{}(id: {}) {:.2f} GB/s {:d} ms", FUNC_NAME(l2_store_kernel), args->id, bandwidth, (int)elapsed_ms);
  
  // cleanup
  CUDA_CALL(hipFree(d_out));
  CUDA_CALL(hipEventDestroy(start));
  CUDA_CALL(hipEventDestroy(end));
}